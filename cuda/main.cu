#include <iostream>
#include <vector>
#include <chrono>
#include "layer.cuh"
#include "util.cuh"
#include "functions.h"
#include "model.cuh"
#include "loadmnist.h"
#include "error_check.cuh"

int main(int argc, char** argv){

    if (argc < 2){
        std::cout << "Usage: " << argv[0] << " <GPU Inference Batch Size>" << std::endl;
        return -1;
    }

    const int num_train_images = 60000;
    const int num_test_images = 10000;
    const int image_width = 28;
    const int image_height = 28;

    //load in train and test images and labels
    float** train_images = readMnistImages("../MNIST/", true);
    int* train_labels = readMnistLabels("../MNIST/", true);
    float** test_images = readMnistImages("../MNIST/", false);
    int* test_labels = readMnistLabels("../MNIST/", false);


    //offload testing images and labels to GPU global memory
    //labels are a 1d array, simple 1d malloc and memcpy
    int* d_test_labels;
    CUDA_CHECK(hipMalloc(&d_test_labels, num_test_images*sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_test_labels, test_labels, num_test_images* sizeof(int), hipMemcpyHostToDevice));
    //1 image is treated as a 1d array and there are 10000 testing images
 
    float* d_test_images;
    CUDA_CHECK(hipMalloc(&d_test_images, num_test_images*image_height*image_width*sizeof(float)));
    for (int i = 0; i < num_test_images; ++i){
        CUDA_CHECK(hipMemcpy(d_test_images + i * image_height * image_width, 
                                test_images[i], image_height * image_width * sizeof(float), hipMemcpyHostToDevice));
    }


    auto relu = ReLU<float>;
    auto softmax = Softmax<float>;


    hyperparams p = {1, 32, 0.01};

    int infer_batch_size = atoi(argv[1]);
    std::cout<<"GPU Inference Batch Size: "<<infer_batch_size<<std::endl;
    Model<float> model = Model<float>(relu, softmax, infer_batch_size);

    for(int i = 0; i < 10; ++i){
        std::cout<<"---Epoch "<<i<<" ---"<<std::endl;
        auto total_start = std::chrono::high_resolution_clock::now();
        //model.test(test_images, image_height*image_width, num_test_images, test_labels);
        int correct = model.gpuInference(d_test_images, d_test_labels, num_test_images, image_height*image_width);
        auto test_end = std::chrono::high_resolution_clock::now();
        std::cout<<correct<<"/"<<num_test_images<<std::endl;
        std::cout<<"Test time: "<<std::chrono::duration_cast<std::chrono::milliseconds>(test_end - total_start).count()<<" ms"<<std::endl;
        model.train(train_images, image_height*image_width, num_train_images, train_labels, p);
        auto train_end = std::chrono::high_resolution_clock::now();
        std::cout<<"Epoch Train time: "<<std::chrono::duration_cast<std::chrono::milliseconds>(train_end - test_end).count()<<" ms"<<std::endl;
        model.updateGPUParams();
    }

    auto total_start = std::chrono::high_resolution_clock::now();
    model.test(test_images, image_height*image_width, num_test_images, test_labels);
    auto test_end = std::chrono::high_resolution_clock::now();
    std::cout<<"Test time: "<<std::chrono::duration_cast<std::chrono::milliseconds>(test_end - total_start).count()<<" ms"<<std::endl;

    //delete test and train images
    for (int i = 0; i < num_train_images; ++i){
        delete[] train_images[i];
    }
    for(int i = 0; i < num_test_images; ++i){
        delete[] test_images[i];
    }
    delete[] train_images;
    delete[] test_images;
    delete[] train_labels;
    delete[] test_labels;

    CUDA_CHECK(hipFree(d_test_labels));
    CUDA_CHECK(hipFree(d_test_images));

    return 0;
}